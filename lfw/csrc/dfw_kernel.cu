#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

// TODO: These constants may be device-dependent
// Maximum threads per block
const unsigned int MAX_TPB = 1024;
// Maximum thread for 1st dim of block
const unsigned int MAX_K_TPB = 1024;
// Maximum thread for 2nd dim of block
const unsigned int MAX_D_TPB = 1024;

#define ceil_div(a, b) (((a) + (b)-1) / (b))

template <typename scalar_t>
__global__ void lfw_cuda_fwd_kernel(
    const scalar_t *query,
    const scalar_t *key,
    const scalar_t *value,
    const scalar_t *state,
    scalar_t *final_state,
    scalar_t *outputs,
    int b_size, int l_size, int d_size, int m_size)
{
    // TODO: this is 0 for now
    int m = 0; // blockDim.x * blockIdx.x + threadIdx.x;
    int d = blockDim.y * blockIdx.y + threadIdx.y;
    int b = blockDim.z * blockIdx.z + threadIdx.z;

    // Holds state (for this specific dimension d)
    // Dynamic shared memory
    extern __shared__ char smem[];
    scalar_t *shared_kv = reinterpret_cast<scalar_t *>(smem);

    // Check bounds
    // TODO: Check m bounds
    if (b < b_size && d < d_size)
    {
        // b, d, m
        int state_offset = (b * d_size + d) * m_size + m;
        // b, t, d, where t = 0
        int d_offset = (b * l_size) * d_size + d;
        // b, t, m, where t = 0
        int k_offset = (b * l_size) * m_size + m;

        // scalar_t cur_s = state[state_offset];
        // Load current state
        for (int m_local = 0; m_local < m_size; m_local++)
        {
            shared_kv[m_local] = state[state_offset + m_local];
        }

        // Go over each time step
        for (int t = 0; t < l_size; t++)
        {
            // Compute next state
            scalar_t out = 0;
            for (int m_local = 0; m_local < m_size; m_local++)
            {
                // Add new value to state
                shared_kv[m_local] += value[d_offset] * key[k_offset + m_local];
                // Query the state
                out += shared_kv[m_local] * query[k_offset + m_local];
            }

            // atomicAdd(
            //     &outputs[d_offset],
            //     out);
            // TODO: Won't parallelize
            outputs[d_offset] = out;

            d_offset += d_size;
            k_offset += m_size;
        }

        // Store final state

        // Load current state
        for (int m_local = 0; m_local < m_size; m_local++)
        {
            final_state[state_offset + m_local] = shared_kv[m_local];
        }
    }
}

// template <typename scalar_t>
// __global__ void lfw_cuda_bwd_kernel(
//     const scalar_t *grad_output, const scalar_t *grad_state,
//     const scalar_t *f, const scalar_t *query,
//     const scalar_t *f_key, const scalar_t *outputs,
//     scalar_t *s_grad, scalar_t *d_state,
//     int b_size, int l_size, int d_size, int k_size)
// {
//     int k = blockDim.x * blockIdx.x + threadIdx.x;
//     int d = blockDim.y * blockIdx.y + threadIdx.y;
//     int b = blockDim.z * blockIdx.z + threadIdx.z;

//     // Check bounds
//     if (b < b_size && d < d_size && k < k_size)
//     {
//         // b, l, d, k
//         const int maxT = l_size - 1;
//         const int state_flat_offset = (b * d_size + d) * k_size + k;
//         int state_offset = ((b * l_size + maxT) * d_size + d) * k_size + k;
//         // b, t, d, where t = max
//         int d_offset = (b * l_size + maxT) * d_size + d;
//         // b, t, k, where t = max
//         int k_offset = (b * l_size + maxT) * k_size + k;

//         auto cur_s_grad = grad_state[state_flat_offset];

//         for (int t = 0; t < l_size; t++)
//         {
//             cur_s_grad = grad_output[d_offset] * query[k_offset] + cur_s_grad;
//             s_grad[state_offset] = cur_s_grad;

//             // Compute next state
//             // Apply current f to gradient
//             cur_s_grad *= f[d_offset] * f_key[k_offset];

//             state_offset -= d_size * k_size;
//             d_offset -= d_size;
//             k_offset -= k_size;
//         }
//         d_state[state_flat_offset] = cur_s_grad;
//     }
// }

// Compute power of two greater than or equal to `n`
unsigned int nextPowerOf2(unsigned int n)
{
    unsigned count = 0;

    // First n in the below condition
    // is for the case where n is 0
    if (n && !(n & (n - 1)))
        return n;

    while (n != 0)
    {
        n >>= 1;
        count += 1;
    }

    return 1 << count;
}

// CUDA declarations
std::vector<torch::Tensor> lfw_cuda_forward(
    torch::Tensor query,
    torch::Tensor key,
    torch::Tensor value,
    torch::Tensor state)
{
    // Length
    const auto L = value.size(1);
    // Batch
    const auto B = state.size(0);
    // Dimension
    const auto D = state.size(1);
    // Expansion dimension
    const auto M = state.size(2);

    // TODO: to save memory could we write back into the same state?
    auto final_state = torch::empty(
        {B, D, M},
        torch::TensorOptions()
            .dtype(value.dtype())
            .device(value.device()));

    auto outputs = torch::empty(
        {B, L, D},
        torch::TensorOptions()
            .dtype(value.dtype())
            .device(value.device()));

    const auto numMThreads = 1;
    // std::min(nextPowerOf2(K), MAX_K_TPB);
    const auto numDThreads = 1;
    // std::min(
    //     nextPowerOf2(D),
    //     std::min(MAX_TPB / numMThreads, MAX_D_TPB));

    const dim3 threads(numMThreads, numDThreads, 1);
    const dim3 blocks(
        1,
        // ceil_div(M, threads.x),
        ceil_div(D, threads.y),
        B);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        value.scalar_type(),
        "lfw_cuda_fwd_kernel",
        ([&]
         { lfw_cuda_fwd_kernel<scalar_t><<<blocks, threads, M * sizeof(scalar_t)>>>(
               query.data<scalar_t>(),
               key.data<scalar_t>(),
               value.data<scalar_t>(),
               state.data<scalar_t>(),
               final_state.data<scalar_t>(),
               outputs.data<scalar_t>(),
               B, L, D, M); }));

    return {outputs, final_state};
}

// std::vector<torch::Tensor> lfw_cuda_backward(
//     torch::Tensor grad_output,
//     torch::Tensor grad_state,
//     torch::Tensor query,
//     torch::Tensor outputs)
// {
//     // Length
//     const auto L = query.size(1);
//     // Batch
//     const auto B = grad_state.size(0);
//     // Dimension
//     const auto D = grad_state.size(1);
//     // Expansion dimension
//     const auto K = grad_state.size(2);

//     // Kernel outputs
//     auto s_grad = torch::empty(
//         {B, L, D, K},
//         torch::TensorOptions()
//             .dtype(grad_state.dtype())
//             .device(grad_state.device()));
//     auto d_state = torch::empty(
//         {B, D, K},
//         torch::TensorOptions()
//             .dtype(grad_state.dtype())
//             .device(grad_state.device()));

//     const auto numMThreads = std::min(nextPowerOf2(K), MAX_K_TPB);
//     const auto numDThreads = std::min(
//         nextPowerOf2(D),
//         std::min(MAX_TPB / numMThreads, MAX_D_TPB));

//     const dim3 threads(numMThreads, numDThreads, 1);
//     const dim3 blocks(
//         ceil_div(K, threads.x),
//         ceil_div(D, threads.y),
//         B);

//     AT_DISPATCH_FLOATING_TYPES_AND_HALF(
//         grad_state.scalar_type(),
//         "lfw_cuda_bwd_kernel",
//         ([&]
//          { lfw_cuda_bwd_kernel<scalar_t><<<blocks, threads>>>(
//                grad_output.data<scalar_t>(),
//                grad_state.data<scalar_t>(),
//                query.data<scalar_t>(),
//                outputs.data<scalar_t>(),
//                s_grad.data<scalar_t>(),
//                d_state.data<scalar_t>(),
//                B, L, D, K); }));
//     return {s_grad, d_state};
// }
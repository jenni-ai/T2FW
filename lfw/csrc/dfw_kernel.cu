#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>
#include <cmath>

// TODO: These constants may be device-dependent
// Maximum threads per block
const unsigned int MAX_TPB = 1024;
// Maximum thread for 1st dim of block
const unsigned int MAX_X_TPB = MAX_TPB;

#define WARP_SIZE 32
#define ceil_div(a, b) (((a) + (b)-1) / (b))

template <uint size, typename scalar_t>
struct Vec
{
    scalar_t vec[size];

    //     auto &operator+=(const Vec<size, scalar_t> *other)
    //     {
    // #pragma unroll
    //         for (int i = 0; i < size; i++)
    //         {
    //             this.vec[i] += other.vec[i];
    //         }
    //         return *this;
    //     }
};

/**
 * @brief Performs reduction within one active warp.
 */
template <typename scalar_t>
__device__ void warpReduce(volatile scalar_t *sdata, int tid)
{
    sdata[tid] += sdata[tid + 32];
    sdata[tid] += sdata[tid + 16];
    sdata[tid] += sdata[tid + 8];
    sdata[tid] += sdata[tid + 4];
    sdata[tid] += sdata[tid + 2];
    sdata[tid] += sdata[tid + 1];
}

/**
 * @brief Performs parallel sum of an array, leaving the result in index 0.
 * Only supports EVEN sizes for correctness.
 *
 * @tparam scalar_t
 * @param arr
 * @param id
 * @param size
 * @return __device__
 */
template <uint width, typename VecType>
__device__ VecType sumReduc(
    VecType *arr,
    const int tid,
    const int size)
{
    for (unsigned int step_size = size / 2; step_size > WARP_SIZE; step_size >>= 1)
    {
        if (tid < step_size)
        {

#pragma unroll
            for (int i = 0; i < width; i++)
            {
                // Reduce to the left side
                arr[tid].vec[i] += arr[tid + step_size].vec[i];
            }
            // arr[tid] += arr[tid + step_size];
        }
        __syncthreads();
    }
    // auto res = arr[0];
    // if (tid < 32)
    //     warpReduce(arr, tid);

    //     VecType res;
    // #pragma unroll
    //     for (int w = 0; w < width; w++)
    //     {
    //         res.vec[w] = 0;
    //     }
    auto res = arr[0];
    for (int i = 1; i < min(WARP_SIZE, size); i++)
    {
#pragma unroll
        for (int w = 0; w < width; w++)
        {
            res.vec[w] += arr[i].vec[w];
        }
    }
    __syncthreads();
    return res;
}

/**
 * @brief Runs each dimension d in parallel.
 */
template <typename scalar_t>
__global__ void lfw_cuda_fwd_kernel(
    const scalar_t *query,
    const scalar_t *key,
    const scalar_t *value,
    const scalar_t *state,
    scalar_t *final_state,
    scalar_t *outputs,
    scalar_t *delta_value,
    int b_size,
    int l_size,
    int d_size,
    int m_size,
    int num_tiles,
    int tile_size)
{
    const int tile_id = blockDim.x * blockIdx.x + threadIdx.x;
    const int d = blockDim.y * blockIdx.y + threadIdx.y;
    const int b = blockDim.z * blockIdx.z + threadIdx.z;

    // Dynamic shared memory
    extern __shared__ char smem[];
    // Holds state (for this specific dimension d) (size = m_size)
    scalar_t *cur_state = reinterpret_cast<scalar_t *>(smem);
    // Holding tile results
    Vec<2, scalar_t> *shared_tile = reinterpret_cast<Vec<2, scalar_t> *>(&cur_state[m_size]);

    // NOTE: Shouldn't be possible to be out of bounds for (b and d)

    // We will be looping from m_start to m_end (which is the size of a tile)
    const int m_start = tile_id * tile_size;
    const int m_end = m_start + tile_size;

    // b, d, m, m = 0
    int state_offset = (b * d_size + d) * m_size;
    // b, t, d, where t = 0
    int d_offset = (b * l_size) * d_size + d;
    // b, t, m, where t = 0, m = 0
    int m_offset = (b * l_size) * m_size;

    // Load current state
    for (int m = m_start; m < m_end && m < m_size; m++)
    {
        cur_state[m] = state[state_offset + m];
    }

    // Go over each time step
    for (int t = 0; t < l_size; t++)
    {
        scalar_t q_out = 0;
        scalar_t k_out = 0;

        for (int m = m_start; m < m_end && m < m_size; m++)
        {
            // Query the state
            q_out += cur_state[m] * query[m_offset + m];
            // Query the old value associated with key
            k_out += cur_state[m] * key[m_offset + m];
        }
        // Each tile produce its partial results
        shared_tile[tile_id].vec[0] = q_out;
        shared_tile[tile_id].vec[1] = k_out;
        __syncthreads();
        auto res = sumReduc<2>(shared_tile, tile_id, num_tiles);
        q_out = res.vec[0];
        k_out = res.vec[1];

        // Write output
        outputs[d_offset] = q_out;

        // Compute the delta
        auto curVal = value[d_offset] - k_out;
        delta_value[d_offset] = curVal;

        // Compute next state
        // TODO: Could merge this loop?
        for (int m = m_start; m < m_end && m < m_size; m++)
        {
            // Add new value to state
            cur_state[m] += curVal * key[m_offset + m];
        }

        d_offset += d_size;
        m_offset += m_size;
    }

    // Store final state
    for (int m = m_start; m < m_end && m < m_size; m++)
    {
        final_state[state_offset + m] = cur_state[m];
    }
}

/**
 * @brief Computes the gradient of d_value, d_state.
 * Runs each dimension d in parallel.
 * Tiles along dimension m.
 */
template <typename scalar_t>
__global__ void lfw_cuda_bwd_value_kernel(
    const scalar_t *grad_output,
    const scalar_t *grad_state,
    const scalar_t *query,
    const scalar_t *key,
    scalar_t *d_value,
    scalar_t *d_state,
    int b_size,
    int l_size,
    int d_size,
    int m_size,
    int num_tiles,
    int tile_size)
{
    const int tile_id = blockDim.x * blockIdx.x + threadIdx.x;
    const int d = blockDim.y * blockIdx.y + threadIdx.y;
    const int b = blockDim.z * blockIdx.z + threadIdx.z;

    // Dynamic shared memory
    extern __shared__ char smem[];
    // Holds recursive gradient of states (for this specific dimension d)
    scalar_t *cur_s_grad = reinterpret_cast<scalar_t *>(smem);
    // Holds tile results (size = num_tiles)
    Vec<1, scalar_t> *shared_tile = reinterpret_cast<Vec<1, scalar_t> *>(&cur_s_grad[d_size]);

    // NOTE: Shouldn't be possible to be out of bounds for (b and d)

    // We will be looping from m_start to m_end (which is the size of a tile)
    const int m_start = tile_id * tile_size;
    const int m_end = m_start + tile_size;

    const int maxT = l_size - 1;
    // b, d, m = 0
    const int state_offset = (b * d_size + d) * m_size;
    // b, t, d, where t = max, d=0
    int d_offset = (b * l_size + maxT) * d_size + d;
    // b, t, m, where t = max, m = 0
    int m_offset = (b * l_size + maxT) * m_size;

    for (int m = m_start; m < m_end && m < m_size; m++)
    {
        // Load final state's gradient
        cur_s_grad[m] = grad_state[state_offset + m];
    }

    // Loops from final timestep to first timestep
    for (int t = 0; t < l_size; t++)
    {
        scalar_t d_v = 0;
        for (int m = m_start; m < m_end && m < m_size; m++)
        {
            // Compute s_grad * k
            d_v += cur_s_grad[m] * key[m_offset + m];
        }

        shared_tile[tile_id].vec[0] = d_v;
        __syncthreads();
        auto res = sumReduc<1>(shared_tile, tile_id, num_tiles);
        d_v = res.vec[0];

        d_value[d_offset] = d_v;

        // Apply delta rule derivatives
        auto g_out = grad_output[d_offset];
        for (int m = m_start; m < m_end && m < m_size; m++)
        {
            auto change = g_out * query[m_offset + m] - d_v * key[m_offset + m];
            cur_s_grad[m] += change;
        }

        d_offset -= d_size;
        m_offset -= m_size;
    }

    // Store d_state
    for (int m = m_start; m < m_end && m < m_size; m++)
    {
        d_state[state_offset + m] = cur_s_grad[m];
    }
}

/**
 * @brief Computes the gradient of d_query and d_key.
 * Runs each dimension m in parallel.
 * Tiles along dimension d.
 */
template <typename scalar_t>
__global__ void lfw_cuda_bwd_qk_kernel(
    const scalar_t *grad_output,
    const scalar_t *grad_state,
    const scalar_t *query,
    const scalar_t *key,
    const scalar_t *delta_value,
    const scalar_t *final_state,
    scalar_t *d_query,
    scalar_t *d_key,
    const scalar_t *d_value,
    int b_size,
    int l_size,
    int d_size,
    int m_size,
    int num_tiles,
    int tile_size)
{
    const int tile_id = blockDim.x * blockIdx.x + threadIdx.x;
    const int m = blockDim.y * blockIdx.y + threadIdx.y;
    const int b = blockDim.z * blockIdx.z + threadIdx.z;

    // Dynamic shared memory
    extern __shared__ char smem[];
    // Holds state (for this specific dimension m)
    scalar_t *cur_state = reinterpret_cast<scalar_t *>(smem);
    // Holds d_state results
    scalar_t *cur_s_grad = &cur_state[d_size];
    // Holds tile results (size = num_tiles)
    Vec<2, scalar_t> *shared_tile = reinterpret_cast<Vec<2, scalar_t> *>(&cur_s_grad[d_size]);

    // NOTE: Shouldn't be possible to be out of bounds for (b and d)

    // We will be looping from m_start to m_end (which is the size of a tile)
    const int d_start = tile_id * tile_size;
    const int d_end = d_start + tile_size;
    const int width = 2;

    const int maxT = l_size - 1;
    // b, d, m, where d = 0
    const int state_offset = (b * d_size) * m_size + m;
    // b, t, d, where t = max, d=0
    int d_offset = (b * l_size + maxT) * d_size;
    // b, t, m, where t = max
    int m_offset = (b * l_size + maxT) * m_size + m;

    for (int d = d_start; d < d_end && d < d_size; d++)
    {
        // Load final state
        cur_state[d] = final_state[state_offset + d * m_size];
        // Load final state's gradient
        cur_s_grad[d] = grad_state[state_offset + d * m_size];
    }

    // Loops from final timestep to first timestep
    for (int t = 0; t < l_size; t++)
    {
        const auto q = query[m_offset];
        const auto k = key[m_offset];

        scalar_t d_q = 0;
        scalar_t d_k = 0;
        for (int d = d_start; d < d_end && d < d_size; d++)
        {
            auto deltaVal = delta_value[d_offset + d];
            // Move state backwards
            cur_state[d] -= deltaVal * k;

            d_q += grad_output[d_offset + d] * cur_state[d];

            // v * s_grad
            d_k += deltaVal * cur_s_grad[d];
            d_k -= d_value[d_offset + d] * cur_state[d];
        }
        shared_tile[tile_id].vec[0] = d_q;
        shared_tile[tile_id].vec[1] = d_k;
        __syncthreads();
        auto res = sumReduc<2>(shared_tile, tile_id, num_tiles);
        d_q = res.vec[0];
        d_k = res.vec[1];

        d_query[m_offset] = d_q;
        d_key[m_offset] = d_k;

        // Apply delta rule derivatives
        for (int d = d_start; d < d_end && d < d_size; d++)
        {
            auto change = grad_output[d_offset + d] * q - d_value[d_offset + d] * k;
            cur_s_grad[d] += change;
        }

        d_offset -= d_size;
        m_offset -= m_size;
    }
}

// Compute power of two greater than or equal to `n`
unsigned int nextPowerOf2(unsigned int n)
{
    unsigned count = 0;

    // First n in the below condition
    // is for the case where n is 0
    if (n && !(n & (n - 1)))
        return n;

    while (n != 0)
    {
        n >>= 1;
        count += 1;
    }

    return 1 << count;
}

// CUDA declarations
std::vector<torch::Tensor> lfw_cuda_forward(
    torch::Tensor query,
    torch::Tensor key,
    torch::Tensor value,
    torch::Tensor state)
{
    // Length
    const auto L = value.size(1);
    // Batch
    const auto B = state.size(0);
    // Dimension
    const auto D = state.size(1);
    // Expansion dimension
    const auto M = state.size(2);

    // TODO: to save memory could we write back into the same state?
    auto final_state = torch::empty(
        {B, D, M},
        torch::TensorOptions()
            .dtype(value.dtype())
            .device(value.device()));

    auto outputs = torch::empty(
        {B, L, D},
        torch::TensorOptions()
            .dtype(value.dtype())
            .device(value.device()));

    auto delta_value = torch::empty(
        {B, L, D},
        torch::TensorOptions()
            .dtype(value.dtype())
            .device(value.device()));

    // const auto max_tiles = std::min((uint)(std::sqrt(M) * 2), MAX_X_TPB);
    const auto num_tiles = nextPowerOf2(std::min((uint)(std::sqrt(M)), MAX_X_TPB));
    // const auto num_tiles = nextPowerOf2(std::min((uint)ceil_div(M, 2), MAX_X_TPB));
    // Elements to process per tile. Must be powers of 2.
    const auto tile_size = ceil_div(M, num_tiles);
    // std::min(
    //     nextPowerOf2(D),
    //     std::min(MAX_TPB / num_tiles, MAX_D_TPB));
    std::cout << "num_tiles: ";
    std::cout << num_tiles;
    std::cout << "\nTile size: ";
    std::cout << tile_size;
    std::cout << "\n";
    // Cannot use same sm for different dims
    const dim3 threads(num_tiles, 1, 1);
    const dim3 blocks(1, D, B);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        value.scalar_type(),
        "lfw_cuda_fwd_kernel",
        ([&]
         { lfw_cuda_fwd_kernel<scalar_t><<<blocks, threads, M * sizeof(scalar_t) + (num_tiles) * sizeof(Vec<2, scalar_t>)>>>(
               query.data<scalar_t>(),
               key.data<scalar_t>(),
               value.data<scalar_t>(),
               state.data<scalar_t>(),
               final_state.data<scalar_t>(),
               outputs.data<scalar_t>(),
               delta_value.data<scalar_t>(),
               B, L, D, M, num_tiles, tile_size); }));

    return {outputs, final_state, delta_value};
}

std::vector<torch::Tensor> lfw_cuda_backward(
    torch::Tensor grad_output,
    torch::Tensor grad_state,
    torch::Tensor query,
    torch::Tensor key,
    torch::Tensor delta_value,
    torch::Tensor final_state)
{
    // Length
    const auto L = query.size(1);
    // Batch
    const auto B = grad_state.size(0);
    // Dimension
    const auto D = grad_state.size(1);
    // Expansion dimension
    const auto M = grad_state.size(2);

    // Kernel outputs
    auto d_query = torch::empty(
        {B, L, M},
        torch::TensorOptions()
            .dtype(grad_state.dtype())
            .device(grad_state.device()));
    auto d_key = torch::empty(
        {B, L, M},
        torch::TensorOptions()
            .dtype(grad_state.dtype())
            .device(grad_state.device()));
    auto d_value = torch::empty(
        {B, L, D},
        torch::TensorOptions()
            .dtype(grad_state.dtype())
            .device(grad_state.device()));
    auto d_state = torch::empty(
        {B, D, M},
        torch::TensorOptions()
            .dtype(grad_state.dtype())
            .device(grad_state.device()));

    // TODO: Would be more efficient to pack rest of dimension into same SM
    auto num_tiles = nextPowerOf2(std::min((uint)(std::sqrt(M)), MAX_X_TPB));
    // Elements to process per tile
    auto tile_size = ceil_div(M, num_tiles);
    // Cannot use same sm for different dims
    dim3 threads(num_tiles, 1, 1);
    dim3 blocks(1, D, B);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        grad_state.scalar_type(),
        "lfw_cuda_bwd_value_kernel",
        ([&]
         { lfw_cuda_bwd_value_kernel<scalar_t><<<blocks, threads, M * sizeof(scalar_t) + num_tiles * sizeof(Vec<1, scalar_t>)>>>(
               grad_output.data<scalar_t>(),
               grad_state.data<scalar_t>(),
               query.data<scalar_t>(),
               key.data<scalar_t>(),
               // Outputs
               d_value.data<scalar_t>(),
               d_state.data<scalar_t>(),
               B, L, D, M, num_tiles, tile_size); }));

    num_tiles = nextPowerOf2(std::min((uint)(std::sqrt(D)), MAX_X_TPB));
    // Elements to process per tile
    tile_size = ceil_div(D, num_tiles);
    // Cannot use same sm for different dims
    dim3 threads_qk(num_tiles, 1, 1);
    dim3 blocks_qk(1, M, B);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        grad_state.scalar_type(),
        "lfw_cuda_bwd_qk_kernel",
        ([&]
         { lfw_cuda_bwd_qk_kernel<scalar_t><<<blocks_qk, threads_qk, (D * 2) * sizeof(scalar_t) + num_tiles * sizeof(Vec<2, scalar_t>)>>>(
               grad_output.data<scalar_t>(),
               grad_state.data<scalar_t>(),
               query.data<scalar_t>(),
               key.data<scalar_t>(),
               delta_value.data<scalar_t>(),
               final_state.data<scalar_t>(),
               // Outputs
               d_query.data<scalar_t>(),
               d_key.data<scalar_t>(),
               d_value.data<scalar_t>(),
               B, L, D, M, num_tiles, tile_size); }));

    return {d_query, d_key, d_value, d_state};
}
#include "hip/hip_runtime.h"
#include <torch/extension.h>

#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include <vector>

// TODO: These constants may be device-dependent
// Maximum threads per block
const unsigned int MAX_TPB = 1024;
// Maximum thread for 1st dim of block
// TODO: Should be dynamic
const unsigned int MAX_TILES = 16;
// const unsigned int MAX_TILES = 1024;
// Maximum thread for 2nd dim of block
const unsigned int MAX_D_TPB = 1024;

#define ceil_div(a, b) (((a) + (b)-1) / (b))

/**
 * @brief Performs parallel sum of an array, leaving the result in index 0.
 *
 * @tparam scalar_t
 * @param shared_tile
 * @param id
 * @param size
 * @return __device__
 */
template <typename scalar_t>
__device__ scalar_t parallel_sum(
    scalar_t *shared_tile,
    int id,
    int size)
{
    int step_size = size / 2;
    while (step_size > 0)
    {
        if (id < step_size)
        {
            // Reduce to the left side
            shared_tile[id] += shared_tile[id + step_size];
        }
        __syncthreads();
        step_size = step_size / 2;
    }
    auto res = shared_tile[0];
    __syncthreads();
    return res;
}

/**
 * @brief Runs each dimension d in parallel.
 */
template <typename scalar_t>
__global__ void lfw_cuda_fwd_kernel(
    const scalar_t *query,
    const scalar_t *key,
    const scalar_t *value,
    const scalar_t *state,
    scalar_t *final_state,
    scalar_t *outputs,
    int b_size,
    int l_size,
    int d_size,
    int m_size,
    int num_tiles,
    int tile_size)
{
    const int tile_id = blockDim.x * blockIdx.x + threadIdx.x;
    const int d = blockDim.y * blockIdx.y + threadIdx.y;
    const int b = blockDim.z * blockIdx.z + threadIdx.z;

    // Dynamic shared memory
    extern __shared__ char smem[];
    // Holds state (for this specific dimension d) (size = m_size)
    scalar_t *cur_state = reinterpret_cast<scalar_t *>(smem);
    // Holds tile results (size = num_tiles)
    scalar_t *shared_tile = &cur_state[m_size];

    // NOTE: Shouldn't be possible to be out of bounds for (b and d)

    // We will be looping from m_start to m_end (which is the size of a tile)
    const int m_start = tile_id * tile_size;
    const int m_end = m_start + tile_size;

    // b, d, m, m = 0
    int state_offset = (b * d_size + d) * m_size;
    // b, t, d, where t = 0
    int d_offset = (b * l_size) * d_size + d;
    // b, t, m, where t = 0, m = 0
    int m_offset = (b * l_size) * m_size;

    // Load current state
    for (int m = m_start; m < m_end && m < m_size; m++)
    {
        cur_state[m] = state[state_offset + m];
    }

    // Go over each time step
    for (int t = 0; t < l_size; t++)
    {
        // TODO: Could load value in SM since it's reused?
        auto curVal = value[d_offset];
        // Compute next state
        scalar_t out = 0;
        for (int m = m_start; m < m_end && m < m_size; m++)
        {
            // Add new value to state
            cur_state[m] += curVal * key[m_offset + m];
            // Query the state
            out += cur_state[m] * query[m_offset + m];
        }

        // Each tile produce its partial results
        shared_tile[tile_id] = out;
        __syncthreads();

        // Sum tile results via parallel reduction
        outputs[d_offset] = parallel_sum(shared_tile, tile_id, num_tiles);

        d_offset += d_size;
        m_offset += m_size;
    }

    // Store final state

    // Load current state
    for (int m = m_start; m < m_end && m < m_size; m++)
    {
        final_state[state_offset + m] = cur_state[m];
    }
}

/**
 * @brief Computes the gradient of query and key.
 * Runs each dimension m in parallel.
 * Tiles along dimension d.
 */
// template <typename scalar_t>
// __global__ void lfw_cuda_bwd_key_kernel(
//     const scalar_t *grad_output,
//     const scalar_t *grad_state,
//     const scalar_t *query,
//     const scalar_t *key,
//     const scalar_t *value,
//     const scalar_t *outputs,
//     const scalar_t *final_state,
//     scalar_t *d_query,
//     scalar_t *d_key,
//     scalar_t *d_value,
//     scalar_t *d_state,
//     int b_size,
//     int l_size,
//     int d_size,
//     int m_size,
//     int num_tiles,
//     int tile_size)
// {
//     const int tile_id = blockDim.x * blockIdx.x + threadIdx.x;
//     const int m = blockDim.y * blockIdx.y + threadIdx.y;
//     const int b = blockDim.z * blockIdx.z + threadIdx.z;

//     // Dynamic shared memory
//     extern __shared__ char smem[];
//     // Holds state (for this specific dimension d)
//     scalar_t *cur_state = reinterpret_cast<scalar_t *>(smem);
//     // Holds d_state results
//     scalar_t *cur_s_grad = &cur_state[d_size];
//     // Holds tile results (size = num_tiles)
//     scalar_t *shared_tile = &cur_s_grad[d_size];

//     // NOTE: Shouldn't be possible to be out of bounds for (b and d)

//     // We will be looping from m_start to m_end (which is the size of a tile)
//     const int m_start = tile_id * tile_size;
//     const int m_end = m_start + tile_size;

//     const int maxT = l_size - 1;
//     // b, d, m = 0
//     const int state_offset = (b * d_size + d) * m_size;
//     // b, t, d, where t = max, d=0
//     // TODO
//     int d_offset = (b * l_size + maxT) * d_size;
//     //  + d;
//     // b, t, m, where t = max, m = 0
//     int m_offset = (b * l_size + maxT) * m_size;

//     for (int m = m_start; m < m_end && m < m_size; m++)
//     {
//         // Load final state
//         cur_state[m] = final_state[state_offset + m];
//         // Load final state's gradient
//         cur_s_grad[m] = grad_state[state_offset + m];
//     }

//     // Loops from final timestep to first timestep
//     for (int t = 0; t < l_size; t++)
//     {
//         auto curVal = value[d_offset];
//         scalar_t tmp_d_query = 0;
//         scalar_t tmp_d_key = 0;
//         for (int m = m_start; m < m_end && m < m_size; m++)
//         {
//             cur_s_grad[m] = grad_output[d_offset] * query[m_offset + m] + cur_s_grad[m];

//             // d_query = grad_output * state
//             tmp_d_query += grad_output[d_offset] * cur_state[m];

//             // Compute previous state (reversing)
//             cur_state[m] -= curVal * key[m_offset + m];

//             // TODO: Apply delta rule derivative
//             // cur_s_grad[m] *= f[d_offset] * f_key[m_offset];

//             tmp_d_key += curVal * cur_state[m];
//         }

//         // Each tile produce its partial results
//         shared_tile[tile_id] = tmp_d_query;
//         __syncthreads();
//         // Sum tile results via parallel reduction
//         d_query[m_offset] = parallel_sum(shared_tile, tile_id, num_tiles);

//         // Each tile produce its partial results
//         shared_tile[tile_id] = tmp_d_key;
//         __syncthreads();
//         // Sum tile results via parallel reduction
//         d_key[m_offset] = parallel_sum(shared_tile, tile_id, num_tiles);

//         d_offset -= d_size;
//         m_offset -= m_size;
//     }

//     for (int m = m_start; m < m_end && m < m_size; m++)
//     {
//         d_state[state_offset + m] = cur_s_grad[m];
//     }
// }

// Compute power of two greater than or equal to `n`
unsigned int nextPowerOf2(unsigned int n)
{
    unsigned count = 0;

    // First n in the below condition
    // is for the case where n is 0
    if (n && !(n & (n - 1)))
        return n;

    while (n != 0)
    {
        n >>= 1;
        count += 1;
    }

    return 1 << count;
}

// CUDA declarations
std::vector<torch::Tensor> lfw_cuda_forward(
    torch::Tensor query,
    torch::Tensor key,
    torch::Tensor value,
    torch::Tensor state)
{
    // Length
    const auto L = value.size(1);
    // Batch
    const auto B = state.size(0);
    // Dimension
    const auto D = state.size(1);
    // Expansion dimension
    const auto M = state.size(2);

    // TODO: to save memory could we write back into the same state?
    auto final_state = torch::empty(
        {B, D, M},
        torch::TensorOptions()
            .dtype(value.dtype())
            .device(value.device()));

    auto outputs = torch::empty(
        {B, L, D},
        torch::TensorOptions()
            .dtype(value.dtype())
            .device(value.device()));

    // TODO: Maybe optimize for tiles = tile_size?
    // TODO: Test with lower max k tpb
    // TODO: Would be more efficient to pack rest of dimension into same SM
    const auto num_tiles = std::min(nextPowerOf2(M), MAX_TILES);
    // Elements to process per tile
    const auto tile_size = ceil_div(M, num_tiles);
    // std::min(
    //     nextPowerOf2(D),
    //     std::min(MAX_TPB / num_tiles, MAX_D_TPB));

    // Cannot use same sm for different dims
    const dim3 threads(num_tiles, 1, 1);
    const dim3 blocks(
        1,
        // ceil_div(M, threads.x),
        ceil_div(D, threads.y),
        B);

    AT_DISPATCH_FLOATING_TYPES_AND_HALF(
        value.scalar_type(),
        "lfw_cuda_fwd_kernel",
        ([&]
         { lfw_cuda_fwd_kernel<scalar_t><<<blocks, threads, (M + num_tiles) * sizeof(scalar_t)>>>(
               query.data<scalar_t>(),
               key.data<scalar_t>(),
               value.data<scalar_t>(),
               state.data<scalar_t>(),
               final_state.data<scalar_t>(),
               outputs.data<scalar_t>(),
               B, L, D, M, num_tiles, tile_size); }));

    return {outputs, final_state};
}

std::vector<torch::Tensor> lfw_cuda_backward(
    torch::Tensor grad_output,
    torch::Tensor grad_state,
    torch::Tensor query,
    torch::Tensor key,
    torch::Tensor value,
    torch::Tensor outputs,
    torch::Tensor final_state)
{
    // Length
    const auto L = query.size(1);
    // Batch
    const auto B = grad_state.size(0);
    // Dimension
    const auto D = grad_state.size(1);
    // Expansion dimension
    const auto M = grad_state.size(2);

    // Kernel outputs
    auto d_query = torch::empty(
        {B, L, M},
        torch::TensorOptions()
            .dtype(grad_state.dtype())
            .device(grad_state.device()));
    auto d_key = torch::empty(
        {B, L, M},
        torch::TensorOptions()
            .dtype(grad_state.dtype())
            .device(grad_state.device()));
    auto d_value = torch::empty(
        {B, L, M},
        torch::TensorOptions()
            .dtype(grad_state.dtype())
            .device(grad_state.device()));
    auto d_state = torch::empty(
        {B, D, M},
        torch::TensorOptions()
            .dtype(grad_state.dtype())
            .device(grad_state.device()));

    // TODO: Maybe optimize for tiles = tile_size?
    // TODO: Test with lower max k tpb
    // TODO: Would be more efficient to pack rest of dimension into same SM
    const auto num_tiles = std::min(nextPowerOf2(D), MAX_TILES);
    // Elements to process per tile
    const auto tile_size = ceil_div(D, num_tiles);
    // Cannot use same sm for different dims
    const dim3 threads(num_tiles, 1, 1);
    const dim3 blocks(
        1,
        ceil_div(M, threads.y),
        B);

    // AT_DISPATCH_FLOATING_TYPES_AND_HALF(
    //     grad_state.scalar_type(),
    //     "lfw_cuda_bwd_key_kernel",
    //     ([&]
    //      { lfw_cuda_bwd_key_kernel<scalar_t><<<blocks, threads, (D * 2 + num_tiles) * sizeof(scalar_t)>>>(
    //            grad_output.data<scalar_t>(),
    //            grad_state.data<scalar_t>(),
    //            query.data<scalar_t>(),
    //            key.data<scalar_t>(),
    //            value.data<scalar_t>(),
    //            outputs.data<scalar_t>(),
    //            final_state.data<scalar_t>(),
    //            // Outputs
    //            d_query.data<scalar_t>(),
    //            d_key.data<scalar_t>(),
    //            d_value.data<scalar_t>(),
    //            d_state.data<scalar_t>(),
    //            B, L, D, M, num_tiles, tile_size); }));

    return {d_query, d_key, d_value, d_state};
}